
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <sys/time.h>
using namespace std;

const int BLOCKSIZE = 32;

// Global variables to calculate sizes
int *sizes_gpu, *subtotals_gpu, *total_gpu;  // For finding sums

// Stopwatch class
class Stopwatch {
  private:
    timeval initialTime;

  public:
    Stopwatch() {
      reset();
    }

    void reset() {
      gettimeofday(&initialTime, 0);
    }

    float elapsed() const {
      timeval currentTime;
      gettimeofday(&currentTime, 0);

      return (float) (currentTime.tv_sec - initialTime.tv_sec) + ((float) (currentTime.tv_usec - initialTime.tv_usec))/1000000;
    }
};

// Create an TownSet of size n with all false entries
void createTownSet(char *current, int n) {
  char temp[n];
  for (int i=0; i<n; i++)
    temp[i] = 0;
  hipMemcpy(&current, &temp, n, hipMemcpyHostToDevice);
}

// Take a union of current_gpu and with specified townset and store it in current_gpu
__global__ void unionTownSet_gpu(char *current, char *townSets, int i, int n) {
  int threadId = blockIdx.x*blockDim.x + threadIdx.x;
  current[threadId] += townSets[i*n+threadId];
}

void unionTownSet(char *current, char *townSets, int i, int n) {
  unionTownSet_gpu<<<n/BLOCKSIZE, BLOCKSIZE>>>(current, townSets, i, n);
}

// Find the total population in all of the town sets
__global__ void sizeTownSet_gpu(char *current, int *pops, int n, int *sizes) {
  int threadId = blockIdx.x*blockDim.x + threadIdx.x;
  if (current[threadId] == 1)
    sizes[threadId] = pops[threadId];
  else
    sizes[threadId] = 0;
}

__global__ void totals_gpu(int *output, int *input) {
  __shared__ int temp[BLOCKSIZE];
  int threadId = blockIdx.x*blockDim.x + threadIdx.x;
  temp[threadIdx.x] = input[threadId];
  __syncthreads();

  for (int s=blockDim.x/2; s>0; s>>=1) {
    if (threadIdx.x < s)
      temp[threadIdx.x] += temp[threadIdx.x+s];
    __syncthreads();
  }

  if (threadIdx.x == 0)
    output[blockIdx.x] = temp[0];
}

int sizeTownSet(char *current, int *pops, int n) {
  sizeTownSet_gpu<<<n/BLOCKSIZE, BLOCKSIZE>>>(current, pops, n, sizes_gpu);
  totals_gpu<<<n/BLOCKSIZE, BLOCKSIZE>>>(subtotals_gpu, sizes_gpu);  // Calculate subtotals
  totals_gpu<<<n/BLOCKSIZE, BLOCKSIZE>>>(total_gpu, subtotals_gpu);  // Calculate subtotals
  int result;
  hipMemcpy(total_gpu, &result, sizeof(int), hipMemcpyDeviceToHost);
  return result;
}

// Count number of towns in a set
__global__ void countTownSet_gpu(char *current, int n, int *sizes) {
  int threadId = blockIdx.x*blockDim.x + threadIdx.x;
  if (current[threadId] == 1)
    sizes[threadId] = 1;
  else
    sizes[threadId] = 0;
}

int countTownSet(char *current, int n) {
  countTownSet_gpu<<<n/BLOCKSIZE, BLOCKSIZE>>>(current, n, sizes_gpu);
  totals_gpu<<<n/BLOCKSIZE, BLOCKSIZE>>>(subtotals_gpu, sizes_gpu);  // Calculate subtotals
  totals_gpu<<<n/BLOCKSIZE, BLOCKSIZE>>>(total_gpu, subtotals_gpu);  // Calculate subtotals
  int result;
  hipMemcpy(total_gpu, &result, sizeof(int), hipMemcpyDeviceToHost);
  return result;
}

int main(int argc, char **argv) {
  if (argc != 4) {
    cout << "Usage: " << argv[0] << " mode size reps" << endl;
    cout << "  Mode is one of the following:" << endl;
    cout << "    a  test full algorithm" << endl;
    cout << "    u  test taking unions" << endl;
    cout << "    s  test calculating sizes" << endl;
    exit(0);
  }
  int size = atoi(argv[2]);
  int reps = atoi(argv[3]);
  char mode = argv[1][0];

  // Initialize data
  char townSets[size*size];
  for (int i=0; i<size; i++) {
    for (int j=0; j<size; j++) {
      if (rand() < .1*RAND_MAX)
        townSets[i*size+j] = 1;
      else
        townSets[i*size+j] = 0;
    }
    townSets[i*size+i] = 1;
  }
  int pops[size];
  int totalPop = 0;
  for (int i=0; i<size; i++) {
    pops[i] = rand() % 256;
    totalPop += pops[i];
  }

  // Perform the tests
  int rounds, totalRounds;
  Stopwatch timer;

  // Copy the data to the GPU
  char *townSets_gpu;
  int *pops_gpu;
  char *current_gpu;  // Stores working townset
  hipMalloc((void**) &townSets_gpu, size*size);
  hipMemcpy(townSets_gpu, townSets, size*size, hipMemcpyHostToDevice);
  hipMalloc((void**) &pops_gpu, size*sizeof(int));
  hipMemcpy(pops_gpu, pops, size*sizeof(int), hipMemcpyHostToDevice);
  hipMalloc((void**) &current_gpu, size);
  hipMalloc((void**) &sizes_gpu, size);
  hipMalloc((void**) &subtotals_gpu, size/BLOCKSIZE);
  hipMalloc((void**) &total_gpu, 1);

  // Do the simulation
  switch (mode) {
    case 'a':
      totalRounds=0;
      for (int r=0; r<reps; r++) {
        createTownSet(current_gpu, size);
        rounds = 0;
        while (sizeTownSet(current_gpu, pops_gpu, size) <= totalPop/2) {
          unionTownSet(current_gpu, townSets_gpu, rand()%size, size);
          rounds++;
        }
        totalRounds += rounds;
      }
      cout << "Average number of rounds = " << ((float) totalRounds/reps) << endl;
      break;
    case 'u':
      createTownSet(current_gpu, size);
      for (int r=0; r<reps; r++)
        unionTownSet(current_gpu, townSets_gpu, rand()%size, size);
      break;
    case 's':
      int s;
      for (int r=0; r<reps; r++)
        s = sizeTownSet(current_gpu, pops_gpu, size);
      break;
  }
  cout << "Elapsed time " << timer.elapsed() << endl;
}
